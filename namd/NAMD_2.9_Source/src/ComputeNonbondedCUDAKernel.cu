#include "hip/hip_runtime.h"

#include "ComputeNonbondedCUDAKernel.h"
#include <stdio.h>

#ifdef NAMD_CUDA

#ifdef WIN32
//  not supported by nvcc on Windows
// #define __thread __declspec(thread)
#define __thread
#endif

__constant__ unsigned int const_exclusions[MAX_CONST_EXCLUSIONS];

static __thread unsigned int *overflow_exclusions;

#define SET_EXCL(EXCL,BASE,DIFF) \
         (EXCL)[((BASE)+(DIFF))>>5] |= (1<<(((BASE)+(DIFF))&31))

void cuda_bind_exclusions(const unsigned int *t, int n) {

  hipMalloc((void**) &overflow_exclusions, n*sizeof(unsigned int));
  cuda_errcheck("malloc overflow_exclusions");
  hipMemcpy(overflow_exclusions, t,
		n*sizeof(unsigned int), hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to overflow_exclusions");
  int nconst = ( n < MAX_CONST_EXCLUSIONS ? n : MAX_CONST_EXCLUSIONS );
  hipMemcpyToSymbol(HIP_SYMBOL(const_exclusions), t, nconst*sizeof(unsigned int), 0);
  cuda_errcheck("memcpy to const_exclusions");
}


texture<float2, 1, hipReadModeElementType> lj_table;
static __thread int lj_table_size;

void cuda_bind_lj_table(const float2 *t, int _lj_table_size) {
    static __thread float2 *ct;
    static __thread int lj_table_alloc;
    lj_table_size = _lj_table_size;
    if ( ct && lj_table_alloc < lj_table_size ) {
      hipFree(ct);
      cuda_errcheck("freeing lj table");
      ct = 0;
    }
    if ( ! ct ) {
      lj_table_alloc = lj_table_size;
      hipMalloc((void**) &ct, lj_table_size*lj_table_size*sizeof(float2));
      cuda_errcheck("allocating lj table");
    }
    hipMemcpy(ct, t, lj_table_size*lj_table_size*sizeof(float2),
                                            hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to lj table");

    lj_table.normalized = false;
    lj_table.addressMode[0] = hipAddressModeClamp;
    lj_table.filterMode = hipFilterModePoint;

    hipBindTexture((size_t*)0, lj_table, ct,
        lj_table_size*lj_table_size*sizeof(float2));
    cuda_errcheck("binding lj table to texture");
}


texture<float4, 1, hipReadModeElementType> force_table;
texture<float4, 1, hipReadModeElementType> energy_table;

void cuda_bind_force_table(const float4 *t, const float4 *et) {
    static __thread hipArray *ct;
    static __thread hipArray *ect;
    if ( ! ct ) {
      hipMallocArray(&ct, &force_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating force table");
    }
    if ( ! ect ) {
      hipMallocArray(&ect, &energy_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating energy table");
    }
    hipMemcpyToArray(ct, 0, 0, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    // hipMemcpy(ct, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to force table");
    hipMemcpyToArray(ect, 0, 0, et, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to energy table");

    force_table.normalized = true;
    force_table.addressMode[0] = hipAddressModeClamp;
    force_table.addressMode[1] = hipAddressModeClamp;
    force_table.filterMode = hipFilterModeLinear;

    energy_table.normalized = true;
    energy_table.addressMode[0] = hipAddressModeClamp;
    energy_table.addressMode[1] = hipAddressModeClamp;
    energy_table.filterMode = hipFilterModeLinear;

    hipBindTextureToArray(force_table, ct);
    cuda_errcheck("binding force table to texture");

    hipBindTextureToArray(energy_table, ect);
    cuda_errcheck("binding energy table to texture");
}

static __thread int patch_pairs_size;
static __thread patch_pair *patch_pairs;
static __thread float *virial_buffers;  // one per patch pair
static __thread float *slow_virial_buffers;  // one per patch pair

static __thread int block_flags_size;
static __thread unsigned int *block_flags;

static __thread int force_lists_size;
static __thread force_list *force_lists;
static __thread unsigned int *force_list_counters;
static __thread unsigned int *GBIS_P1_counters;
static __thread unsigned int *GBIS_P2_counters;
static __thread unsigned int *GBIS_P3_counters;

static __thread int force_buffers_size;
static __thread float4 *force_buffers;
static __thread float4 *slow_force_buffers;

static __thread int atoms_size;
static __thread atom *atoms;
static __thread atom_param *atom_params;
static __thread float4 *forces;
static __thread float4 *slow_forces;
static __thread float *virials;  // one per patch
static __thread float *slow_virials;  // one per patch
static __thread float *energy_gbis;  // one per patch
static __thread float *energy_gbis_buffers;  // one per pair

//GBIS arrays
static __thread float  *intRad0D;      // one per patch
static __thread float  *intRadSD;      // one per patch
static __thread GBReal *psiSumD;     // one per patch
static __thread GBReal *psiSumD_buffers; // one per patch
static __thread float  *bornRadD;     // one per patch
static __thread GBReal *dEdaSumD;    // one per patch
static __thread GBReal *dEdaSumD_buffers; // one per patch
static __thread float  *dHdrPrefixD;  // one per patch

static __thread int patch_pairs_alloc;
static __thread int block_flags_alloc;
static __thread int force_buffers_alloc;
static __thread int force_lists_alloc;
static __thread int atoms_alloc;

static __thread int max_atoms_per_patch;

__thread hipStream_t stream;
__thread hipStream_t stream2;
 
void cuda_init() {
  forces = 0;
  slow_forces = 0;
  virials = 0;
  energy_gbis = 0;
  slow_virials = 0;
  atom_params = 0;
  atoms = 0;
  force_buffers = 0;
  slow_force_buffers = 0;
  force_lists = 0;
  force_list_counters = 0;
  GBIS_P1_counters = 0;
  GBIS_P2_counters = 0;
  GBIS_P3_counters = 0;
  patch_pairs = 0;
  virial_buffers = 0;
  energy_gbis_buffers = 0;
  slow_virial_buffers = 0;
  block_flags = 0;

  intRad0D = 0;
  intRadSD = 0;
  psiSumD = 0;
  psiSumD_buffers = 0;
  bornRadD = 0;
  dEdaSumD = 0;
  dEdaSumD_buffers = 0;
  dHdrPrefixD = 0;

  patch_pairs_alloc = 0;
  block_flags_alloc = 0;
  force_buffers_alloc = 0;
  force_lists_alloc = 0;
  atoms_alloc = 0;

  hipStreamCreate(&stream);
  hipStreamCreate(&stream2);
  cuda_errcheck("hipStreamCreate");
}

void cuda_bind_patch_pairs(const patch_pair *pp, int npp,
                        const force_list *fl, int nfl,
                        int atoms_size_p, int force_buffers_size_p,
                        int block_flags_size_p, int max_atoms_per_patch_p) {

  patch_pairs_size = npp;
  force_buffers_size = force_buffers_size_p;
  force_lists_size = nfl;
  atoms_size = atoms_size_p;
  block_flags_size = block_flags_size_p;
  max_atoms_per_patch = max_atoms_per_patch_p;

#if 0
 printf("%d %d %d %d %d %d %d %d\n",
      patch_pairs_size , patch_pairs_alloc ,
      force_buffers_size , force_buffers_alloc ,
      force_lists_size , force_lists_alloc ,
      atoms_size , atoms_alloc );
#endif

 if ( patch_pairs_size > patch_pairs_alloc ||
      block_flags_size > block_flags_alloc ||
      force_buffers_size > force_buffers_alloc ||
      force_lists_size > force_lists_alloc ||
      atoms_size > atoms_alloc ) {

  block_flags_alloc = (int) (1.2 * block_flags_size);
  patch_pairs_alloc = (int) (1.2 * patch_pairs_size);
  force_buffers_alloc = (int) (1.2 * force_buffers_size);
  force_lists_alloc = (int) (1.2 * force_lists_size);
  atoms_alloc = (int) (1.2 * atoms_size);

  // if ( forces ) hipFree(forces);
  // if ( slow_forces ) hipFree(slow_forces);
  forces = slow_forces = 0;
  if ( atom_params ) hipFree(atom_params);
  if ( atoms ) hipFree(atoms);
  if ( force_buffers ) hipFree(force_buffers);
  if ( slow_force_buffers ) hipFree(slow_force_buffers);
  if ( force_lists ) hipFree(force_lists);
  if ( force_list_counters ) hipFree(force_list_counters);
  if ( GBIS_P1_counters ) hipFree(GBIS_P1_counters);
  if ( GBIS_P2_counters ) hipFree(GBIS_P2_counters);
  if ( GBIS_P3_counters ) hipFree(GBIS_P3_counters);
  // if ( virials ) hipFree(virials);
  virials = slow_virials = 0;
  energy_gbis = 0;
  if ( patch_pairs ) hipFree(patch_pairs);
  if ( virial_buffers ) hipFree(virial_buffers);
  if ( energy_gbis_buffers ) hipFree(energy_gbis_buffers);
  if ( slow_virial_buffers ) hipFree(slow_virial_buffers);
  if ( block_flags ) hipFree(block_flags);
  if ( intRad0D ) hipFree(intRad0D); // GBIS memory
  if ( intRadSD ) hipFree(intRadSD);
  //if ( psiSumD ) hipFree(psiSumD);
  if ( psiSumD_buffers ) hipFree(psiSumD_buffers);
  if ( bornRadD ) hipFree(bornRadD);
  //if ( dEdaSumD ) hipFree(dEdaSumD);
  if ( dEdaSumD_buffers ) hipFree(dEdaSumD_buffers);
  if ( dHdrPrefixD ) hipFree(dHdrPrefixD);
  cuda_errcheck("free everything");

#if 0
  int totalmem = patch_pairs_alloc * sizeof(patch_pair) +
		force_lists_alloc * sizeof(force_list) +
		2 * force_buffers_alloc * sizeof(float4) +
		atoms_alloc * sizeof(atom) +
		atoms_alloc * sizeof(atom_param) +
		2 * atoms_alloc * sizeof(float4);
  // printf("allocating %d MB of memory on GPU\n", totalmem >> 20);
  printf("allocating %d MB of memory for block flags\n",
				(block_flags_alloc * 4) >> 20);
#endif

  hipMalloc((void**) &block_flags, block_flags_alloc * 4);
  hipMalloc((void**) &energy_gbis_buffers, patch_pairs_alloc * sizeof(float));
  hipMalloc((void**) &virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &slow_virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &patch_pairs, patch_pairs_alloc * sizeof(patch_pair));
  // hipMalloc((void**) &virials, 2 * force_lists_alloc * 16*sizeof(float));
  // slow_virials = virials + force_lists_size * 16;
  hipMalloc((void**) &force_lists, force_lists_alloc * sizeof(force_list));
  hipMalloc((void**) &force_list_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &GBIS_P1_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &GBIS_P2_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &GBIS_P3_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &slow_force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &atoms, atoms_alloc * sizeof(atom));
  hipMalloc((void**) &atom_params, atoms_alloc * sizeof(atom_param));
  // hipMalloc((void**) &forces, atoms_alloc * sizeof(float4));
  // hipMalloc((void**) &slow_forces, atoms_alloc * sizeof(float4));
  hipMalloc((void**) &intRad0D, atoms_alloc * sizeof(float));
  hipMalloc((void**) &intRadSD, atoms_alloc * sizeof(float));
  //hipMalloc((void**) &psiSumD, atoms_alloc * sizeof(GBReal));
  hipMalloc((void**) &psiSumD_buffers, force_buffers_alloc * sizeof(GBReal));
  hipMalloc((void**) &bornRadD, atoms_alloc * sizeof(float));
  //hipMalloc((void**) &dEdaSumD, atoms_alloc * sizeof(GBReal));
  hipMalloc((void**) &dEdaSumD_buffers, force_buffers_alloc * sizeof(GBReal));
  hipMalloc((void**) &dHdrPrefixD, atoms_alloc * sizeof(float));
  cuda_errcheck("malloc everything");

 } //if sizes grew

  hipMemcpy(patch_pairs, pp, npp * sizeof(patch_pair),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to patch_pairs");

  hipMemcpy(force_lists, fl, nfl * sizeof(force_list),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to force_lists");

  hipMemset(force_list_counters, 0, nfl * sizeof(unsigned int));
  hipMemset(GBIS_P1_counters, 0, nfl * sizeof(unsigned int));
  hipMemset(GBIS_P2_counters, 0, nfl * sizeof(unsigned int));
  hipMemset(GBIS_P3_counters, 0, nfl * sizeof(unsigned int));
  cuda_errcheck("memset force_list_counters");
} // bind patch pairs

void cuda_bind_atom_params(const atom_param *t) {
  hipMemcpyAsync(atom_params, t, atoms_size * sizeof(atom_param),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atom_params");
}

void cuda_bind_atoms(const atom *a) {
  cuda_errcheck("before memcpy to atoms");
  hipMemcpyAsync(atoms, a, atoms_size * sizeof(atom),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atoms");
}

void cuda_bind_forces(float4 *f, float4 *f_slow) {
  hipHostGetDevicePointer(&forces, f, 0);
  cuda_errcheck("hipHostGetDevicePointer forces");
  hipHostGetDevicePointer(&slow_forces, f_slow, 0);
  cuda_errcheck("hipHostGetDevicePointer slow_forces");
}

void cuda_bind_virials(float *v) {
  hipHostGetDevicePointer(&virials, v, 0);
  cuda_errcheck("hipHostGetDevicePointer virials");
  slow_virials = virials + force_lists_size*16;
}

//GBIS bindings
void cuda_bind_GBIS_energy(float *e) {
  hipHostGetDevicePointer(&energy_gbis, e, 0);
  cuda_errcheck("hipHostGetDevicePointer energy_gbis");
}
void cuda_bind_GBIS_intRad(float *intRad0H, float *intRadSH) {
  hipMemcpyAsync(intRad0D, intRad0H, atoms_size * sizeof(float),
				hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(intRadSD, intRadSH, atoms_size * sizeof(float),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to intRad");
}

void cuda_bind_GBIS_psiSum(GBReal *psiSumH) {
  hipHostGetDevicePointer(&psiSumD, psiSumH, 0);
  cuda_errcheck("hipHostGetDevicePointer psiSum");
}

void cuda_bind_GBIS_bornRad(float *bornRadH) {
  hipMemcpyAsync(bornRadD, bornRadH, atoms_size * sizeof(float),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to bornRad");
}

void cuda_bind_GBIS_dEdaSum(GBReal *dEdaSumH) {
  hipHostGetDevicePointer(&dEdaSumD, dEdaSumH, 0);
  cuda_errcheck("hipHostGetDevicePointer dEdaSum");
}

void cuda_bind_GBIS_dHdrPrefix(float *dHdrPrefixH) {
  hipMemcpyAsync(dHdrPrefixD, dHdrPrefixH, atoms_size * sizeof(float),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to dHdrPrefix");
}
// end GBIS methods

#if 0
void cuda_load_forces(float4 *f, float4 *f_slow, int begin, int count) {
  // printf("load forces %d %d %d\n",begin,count,atoms_size);
  hipMemcpyAsync(f+begin, forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  if ( f_slow ) {
    hipMemcpyAsync(f_slow+begin, slow_forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  }
  cuda_errcheck("memcpy from forces");
}

void cuda_load_virials(float *v, int doSlow) {
  int count = force_lists_size;
  if ( doSlow ) count *= 2;
  hipMemcpyAsync(v, virials, count * 16*sizeof(float),
				hipMemcpyDeviceToHost, stream);
  cuda_errcheck("memcpy from virials");
}
#endif

#if 0
__host__ __device__ static int3 patch_coords_from_id(
        dim3 PATCH_GRID, int id) {

  return make_int3( id % PATCH_GRID.x,
                ( id / PATCH_GRID.x ) % PATCH_GRID.y,
                id / ( PATCH_GRID.x * PATCH_GRID.y ) );
}

__host__ __device__ static int patch_id_from_coords(
        dim3 PATCH_GRID, int3 coords) {

  // handles periodic boundaries
  int x = (coords.x + 4 * PATCH_GRID.x) % PATCH_GRID.x;
  int y = (coords.y + 4 * PATCH_GRID.y) % PATCH_GRID.y;
  int z = (coords.z + 4 * PATCH_GRID.z) % PATCH_GRID.z;

  return ( z * PATCH_GRID.y + y ) * PATCH_GRID.x + x;
}

__host__ __device__ static int3 patch_offset_from_neighbor(int neighbor) {

  // int3 coords = patch_coords_from_id(make_uint3(3,3,3), 13 + neighbor);
  int3 coords = patch_coords_from_id(make_uint3(3,3,3), neighbor);
  return make_int3(coords.x - 1, coords.y - 1, coords.z - 1);

}
#endif
 
#define BLOCK_SIZE 128
#define SHARED_SIZE 32


#define MAKE_PAIRLIST
#define DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef MAKE_PAIRLIST
#define DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"


void cuda_nonbonded_forces(float3 lata, float3 latb, float3 latc,
		float cutoff2, float plcutoff2,
		int cbegin, int ccount, int pbegin, int pcount,
		int doSlow, int doEnergy, int usePairlists, int savePairlists,
		hipStream_t &strm) {

 if ( ccount ) {
   if ( usePairlists ) {
     if ( ! savePairlists ) plcutoff2 = 0.;
   } else {
     plcutoff2 = cutoff2;
   }
   int grid_dim = 65535;  // maximum allowed
   for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
     if ( grid_dim > ccount - cstart ) grid_dim = ccount - cstart;
     // printf("%d %d %d\n",cbegin+cstart,grid_dim,patch_pairs_size);

#define CALL(X) X<<< grid_dim, BLOCK_SIZE, 0, strm \
	>>>(patch_pairs+cbegin+cstart,atoms,atom_params,force_buffers, \
	     (doSlow?slow_force_buffers:0), block_flags, \
             virial_buffers, (doSlow?slow_virial_buffers:0), \
             overflow_exclusions, force_list_counters, force_lists, \
             forces, virials, \
             (doSlow?slow_forces:0), (doSlow?slow_virials:0), \
             lj_table_size, \
	     lata, latb, latc, cutoff2, plcutoff2, doSlow)
//end definition

     if ( doEnergy ) {
       if ( doSlow ) {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_slow_energy_pairlist);
         else CALL(dev_nonbonded_slow_energy);
       } else {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_energy_pairlist);
         else CALL(dev_nonbonded_energy);
       }
     } else {
       if ( doSlow ) {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_slow_pairlist);
         else CALL(dev_nonbonded_slow);
       } else {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_pairlist);
         else CALL(dev_nonbonded);
       }
     }

     cuda_errcheck("dev_nonbonded");
   }
 }

#if 0
 if ( pcount ) {
  // printf("%d %d %d\n",pbegin,pcount,force_lists_size);
  dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,force_buffers,
                virial_buffers,forces,virials);
  if ( doSlow ) {
    dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,slow_force_buffers,
                slow_virial_buffers,slow_forces,slow_virials);
  }
  cuda_errcheck("dev_sum_forces");
 }
#endif

}

//import GBIS Kernel definitions
#include "ComputeGBISCUDAKernel.h"

//////////////////////////////////////////
//  GBIS P1
//////////////////////////////////////////
void cuda_GBIS_P1(
	int cbegin,
  int ccount,
  int pbegin,
  int pcount,
  float a_cut,
  float rho_0,
  float3 lata,
  float3 latb,
  float3 latc,
  hipStream_t &strm
) {

  int grid_dim = 65535;  // maximum allowed
  for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
    if (grid_dim > ccount - cstart) {
      grid_dim = ccount - cstart;
    }

    GBIS_P1_Kernel<<<grid_dim, BLOCK_SIZE, 0, strm>>>(
      patch_pairs+cbegin+cstart,
      atoms,
      atom_params,
      intRad0D,
      intRadSD,
      psiSumD_buffers,
      psiSumD,
      a_cut,
      rho_0,
      lata,
      latb,
      latc,
      force_lists,
      GBIS_P1_counters 
      );
    cuda_errcheck("dev_GBIS_P1");
  } // end for
} // end GBIS P1

//////////////////////////////////////////
//  GBIS P2
//////////////////////////////////////////
void cuda_GBIS_P2(
	int cbegin,
  int ccount,
  int pbegin,
  int pcount,
  float a_cut,
  float r_cut,
  float scaling,
  float kappa,
  float smoothDist,
  float epsilon_p,
  float epsilon_s,
  float3 lata,
  float3 latb,
  float3 latc,
  int doEnergy,
  int doFullElec,
  hipStream_t &strm
) {
  int grid_dim = 65535;  // maximum allowed
  for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
    if (grid_dim > ccount - cstart)
      grid_dim = ccount - cstart;

    GBIS_P2_Kernel<<<grid_dim, BLOCK_SIZE, 0, strm>>>(
      patch_pairs+cbegin+cstart,
      atoms,
      atom_params,
      bornRadD,
      dEdaSumD_buffers,
      dEdaSumD,
      a_cut,
      r_cut,
      scaling,
      kappa,
      smoothDist,
      epsilon_p,
      epsilon_s,
      lata,
      latb,
      latc,
      doEnergy,
      doFullElec,
      force_lists,
      force_buffers,
      forces,
      energy_gbis_buffers,
      energy_gbis,
      GBIS_P2_counters 
      );
    cuda_errcheck("dev_GBIS_P2");
  } // end for
} // end P2

//////////////////////////////////////////
//  GBIS P3
//////////////////////////////////////////
void cuda_GBIS_P3(
	int cbegin,
  int ccount,
  int pbegin,
  int pcount,
  float a_cut,
  float rho_0,
  float scaling,
  float3 lata,
  float3 latb,
  float3 latc,
  hipStream_t &strm
) {
  int grid_dim = 65535;  // maximum allowed
  for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
    if (grid_dim > ccount - cstart)
      grid_dim = ccount - cstart;

    GBIS_P3_Kernel<<<grid_dim, BLOCK_SIZE, 0, strm>>>(
      patch_pairs+cbegin+cstart,
      atoms,
      atom_params,
      intRad0D,
      intRadSD,
      dHdrPrefixD,
      a_cut,
      rho_0,
      scaling,
      lata,
      latb,
      latc,
      force_lists,
      slow_force_buffers,
      slow_forces,
      GBIS_P3_counters 
      );
    cuda_errcheck("dev_GBIS_P3");
  }
}

#if 0
int cuda_stream_finished() {
  return ( hipStreamQuery(stream) == hipSuccess );
}
#endif


#else  // NAMD_CUDA

// for make depends
#include "ComputeNonbondedCUDAKernelBase.h"

#endif  // NAMD_CUDA

